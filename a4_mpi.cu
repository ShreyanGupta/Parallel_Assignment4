#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <cstdlib>
#include <cassert>
#include <cmath>
// #include <string>
#include <hip/hip_runtime.h>
#include <mpi.h>
using namespace std;

// module load compiler/cuda/7.5/compilervars
// module load compiler/gcc/4.9.3/compilervars
// module load mpi/mpich/3.1.4/gcc/mpivars
// module load apps/lammps/gpu

#define thread 1024

int pid;
int k;
int start=0, end, size;
string writefile;
string readfile;

int dim = -1;
vector<int> ptr, indices, data;
vector<int> B;
vector<long long> C;

int *d_dim;
int *d_ptr, *d_indices, *d_data;
int *d_B;
long long *d_C;

void parse_input(){

	// string input in ifstream not working. Don't ask why
	// stoi not working. Also don't ask why
	ifstream fin(readfile.c_str());
	string r,c,d,temp;
	fin >> temp >> temp >> temp >> dim >> temp;
	fin >> r >> c >> d;
	B = vector<int>(dim);
	int current_row = 0;
	int local_ptr = 0;
	ptr.push_back(0);

	while(r[0] != 'B'){
		// cout << r << " " << c << " " << d << endl;
		int this_row = atoi(r.c_str());
		for(int i=current_row; i<this_row; ++i) ptr.push_back(local_ptr);
		indices.push_back(atoi(c.c_str()));
		data.push_back(atoi(d.c_str()));
		current_row = this_row;
		++local_ptr;
		fin >> r >> c >> d;
	}
	ptr.push_back(local_ptr);
	B[0] = atoi(c.c_str());
	B[1] = atoi(d.c_str());
	for(int i=2; i<dim; ++i) fin >> B[i];

	// auto doesn't work for some reason...
	// cout << "dim " << dim << endl;
	// cout << "ptr "; for(int i=0; i<ptr.size(); ++i) cout << ptr[i] << " "; cout << endl;
	// cout << "indices "; for(int i=0; i<indices.size(); ++i) cout << indices[i] << " "; cout << endl;
	// cout << "data "; for(int i=0; i<data.size(); ++i) cout << data[i] << " "; cout << endl;
	// cout << "B "; for(int i=0; i<B.size(); ++i) cout << B[i] << " "; cout << endl;
	// cout << "End of parsing\n";
}

void init(){
	C = vector<long long>(end-start);

	hipMalloc((void **)&d_dim, sizeof(int));
	hipMalloc((void **)&d_ptr, (end-start+1) * sizeof(int));
	hipMalloc((void **)&d_indices, indices.size() * sizeof(int));
	hipMalloc((void **)&d_data, data.size() * sizeof(int));
	hipMalloc((void **)&d_B, B.size() * sizeof(int));
	hipMalloc((void **)&d_C, C.size() * sizeof(long long));

	hipMemcpy(d_dim, &dim, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ptr, &ptr[start], (end-start+1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_indices, &indices[0], indices.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_data, &data[0], data.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, &B[0], B.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C, &C[0], C.size() * sizeof(long long), hipMemcpyHostToDevice);
}

void anti_init(){
	hipMemcpy(&C[0], d_C, C.size() * sizeof(long long), hipMemcpyDeviceToHost);

	hipFree(d_dim);
	hipFree(d_ptr);
	hipFree(d_indices);
	hipFree(d_data);
	hipFree(d_B);
}

__global__ void kernel(
		int *dim, 
		int *ptr, 
		int *indices, 
		int *data,
		int *B,
		long long *C)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x;
	if(row < *dim){
		long long sum = 0;
		for(int i=ptr[row]; i<ptr[row+1]; ++i)
			sum += data[i] * B[indices[i]];
		C[row] = sum;
		// printf("Final row %d sum %d\n", row, sum);
	}
}

__global__ void kernel_complex(
		int *dim, 
		int *ptr, 
		int *indices, 
		int *data,
		int *B,
		long long *C)
{
	__shared__ long long sum[thread];
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = threadIdx.x;
	int row = id/32;
	int lane = id & (32 - 1);
	sum[tid] = 0;
	if(row < *dim){
		// printf("row %d sum %d tid %d lane %d\n", row, sum[tid], tid, lane);
		for(int i=ptr[row] + lane; i<ptr[row+1]; i+=32){
			sum[tid] += (long long)data[i] * (long long)B[indices[i]];
		}
		// printf("row %d sum %d tid %d lane %d\n", row, sum[tid], tid, lane);
	}
	// __syncthreads();
	if(lane < 16) sum[tid] += sum[tid + 16];
	if(lane < 8) sum[tid] += sum[tid + 8];
	if(lane < 4) sum[tid] += sum[tid + 4];
	if(lane < 2) sum[tid] += sum[tid + 2];
	if(lane < 1) sum[tid] += sum[tid + 1];
	if(lane == 0) C[row] = sum[tid];
	// printf("Final row %d sum %d tid %d \n", row, sum[tid], tid);
}

void send_receive_data(){
	if(pid == 0){
		start = 0;
		end = dim/k;
		size = ptr[end] - ptr[start];
		for(int i=1; i<k; ++i){
			// Send indices and data to process i
			int start = i*dim/k;
			int end = (i+1)*dim/k;
			int size = ptr[end] - ptr[start];
			MPI_Send(&indices[ptr[start]], size, MPI_INT, i, 0, MPI_COMM_WORLD);
			MPI_Send(&data[ptr[start]], size, MPI_INT, i, 0, MPI_COMM_WORLD);
		}
		indices.resize(ptr[dim/k]);
		data.resize(ptr[dim/k]);
	}
	else{
		start = pid*dim/k;
		end = (pid+1)*dim/k;
		size = ptr[end] - ptr[start];
		indices = vector<int>(size);
		data = vector<int>(size);
		// printf("pid %d start %d end %d size %d ptr_st %d ptr_end %d\n", pid, start, end, size, ptr[start], ptr[end]);
		MPI_Recv(&indices[0], size, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
		MPI_Recv(&data[0], size, MPI_INT, 0, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);

		for(int i=end; i>=start; --i) ptr[i] -= ptr[start];
	}
}

void write(){
	// Write to file
	for(int j = 0; j < k; ++j) {
		ofstream fout;
		fout.open(writefile.c_str(), std::ios::app);
		if(pid == j) {
			for(int i=0; i<C.size(); ++i) {
				fout << C[i] << "\n";
			}
		}
		fout.close();
		MPI_Barrier(MPI_COMM_WORLD);
	}

	// int next = 1;
	// ofstream fout;
	// if(pid != 0) MPI_Recv(&next, 1, MPI_INT, pid-1, 0, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
	// if(pid != 0) fout.open(writefile.c_str(), std::fstream::app);
	// else fout.open(writefile.c_str());
	// for(int i=0; i<C.size(); ++i) fout << C[i] << "\n";
	// fout.close();
	// if(pid != k-1) MPI_Send(&next, 1, MPI_INT, pid+1, 0, MPI_COMM_WORLD);
}

int main(int argc, char const *argv[])
{
	MPI_Init(NULL, NULL);
	MPI_Comm_rank(MPI_COMM_WORLD, &pid);
	MPI_Comm_size(MPI_COMM_WORLD, &k);

	readfile = string(argv[1]);
	writefile = string(argv[2]);
	if(pid == 0) parse_input();
	MPI_Bcast(&dim, 1, MPI_INT, 0, MPI_COMM_WORLD);
	if(pid != 0) ptr = vector<int>(dim+1);
	if(pid != 0) B = vector<int>(dim);
	MPI_Bcast(&ptr[0], dim+1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&B[0], dim, MPI_INT, 0, MPI_COMM_WORLD);
	send_receive_data();

	// if(pid == 0){
	// 	cout << "pid " << pid << " size " << size << endl;
	// 	cout << "pid " << pid << " dim " << dim << endl;
	// 	cout << "pid " << pid << " ptr "; for(int i=start; i<=end; ++i) cout << ptr[i] << " "; cout << endl;
	// 	cout << "pid " << pid << " indices "; for(int i=0; i<indices.size(); ++i) cout << indices[i] << " "; cout << endl;
	// 	cout << "pid " << pid << " data "; for(int i=0; i<data.size(); ++i) cout << data[i] << " "; cout << endl;
	// 	cout << "pid " << pid << " B "; for(int i=0; i<B.size(); ++i) cout << B[i] << " "; cout << endl;
	// }
	// MPI_Barrier(MPI_COMM_WORLD);
	// if(pid == 1){
	// 	cout << "pid " << pid << " size " << size << endl;
	// 	cout << "pid " << pid << " dim " << dim << endl;
	// 	cout << "pid " << pid << " ptr "; for(int i=start; i<=end; ++i) cout << ptr[i] << " "; cout << endl;
	// 	cout << "pid " << pid << " indices "; for(int i=0; i<indices.size(); ++i) cout << indices[i] << " "; cout << endl;
	// 	cout << "pid " << pid << " data "; for(int i=0; i<data.size(); ++i) cout << data[i] << " "; cout << endl;
	// 	cout << "pid " << pid << " B "; for(int i=0; i<B.size(); ++i) cout << B[i] << " "; cout << endl;
	// }

	init();

	int block = ceil(1.0f * C.size()/(thread/32));
	kernel_complex<<<block, thread>>>(d_dim, d_ptr, d_indices, d_data, d_B, d_C);
	// int block = ceil(1.0f * C.size()/thread);
	// kernel<<<block, thread>>>(d_dim, d_ptr, d_indices, d_data, d_B, d_C);
	cout << "block " << block << " C " << C.size() << endl;

	anti_init();
	
	// cout << "C for pid " << pid << " : "; for(int i=0; i<C.size(); ++i) cout << C[i] << " "; cout << endl;
	write();
	MPI_Finalize();
	return 0;
}