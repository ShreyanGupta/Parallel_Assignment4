#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <cstdlib>
// #include <string>
#include <hip/hip_runtime.h>
using namespace std;

// module load compiler/cuda/7.5/compilervars
// module load compiler/gcc/4.9.3/compilervars
// module load mpi/mpich/3.1.4/gcc/mpivars
// module load apps/lammps/gpu

int dim = -1;
vector<int> ptr, indices, data;
vector<int> B,C;

int *d_dim;
int *d_ptr, *d_indices, *d_data;
int *d_B, *d_C;

void parse_input(string file){

	// string input in ifstream not working. Don't ask why
	// stoi not working. Also don't ask why
	ifstream fin(file.c_str());
	string r,c,d,temp;
	fin >> temp >> temp >> temp >> dim >> temp;
	fin >> r >> c >> d;
	B = vector<int>(dim);
	C = vector<int>(dim);
	int current_row = 0;
	int local_ptr = 0;
	ptr.push_back(0);

	while(r[0] != 'B'){
		// cout << r << " " << c << " " << d << endl;
		int this_row = atoi(r.c_str());
		for(int i=current_row; i<this_row; ++i) ptr.push_back(local_ptr);
		indices.push_back(atoi(c.c_str()));
		data.push_back(atoi(d.c_str()));
		current_row = this_row;
		++local_ptr;
		fin >> r >> c >> d;
	}
	ptr.push_back(local_ptr);
	B[0] = atoi(c.c_str());
	B[1] = atoi(d.c_str());
	for(int i=2; i<dim; ++i) fin >> B[i];

	// auto doesn't work for some reason...
	for(int i=0; i<ptr.size(); ++i) 	cout << ptr[i] << " "; cout << endl;
	for(int i=0; i<indices.size(); ++i) cout << indices[i] << " "; cout << endl;
	for(int i=0; i<data.size(); ++i) 	cout << data[i] << " "; cout << endl;
	for(int i=0; i<B.size(); ++i) 		cout << B[i] << " "; cout << endl;
}

void init(){
	hipMalloc((void **)&d_dim, sizeof(int));
	hipMalloc((void **)&d_ptr, ptr.size() * sizeof(int));
	hipMalloc((void **)&d_indices, indices.size() * sizeof(int));
	hipMalloc((void **)&d_data, data.size() * sizeof(int));
	hipMalloc((void **)&d_B, B.size() * sizeof(int));
	hipMalloc((void **)&d_C, C.size() * sizeof(int));

	hipMemcpy(d_dim, &dim, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_ptr, &ptr[0], ptr.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_indices, &indices[0], indices.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_data, &data[0], data.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_B, &B[0], B.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_C, &C[0], C.size() * sizeof(int), hipMemcpyHostToDevice);
}

void anti_init(){
	hipMemcpy(&C[0], d_C, C.size() * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_dim);
	hipFree(d_ptr);
	hipFree(d_indices);
	hipFree(d_data);
	hipFree(d_B);
}

__global__ void kernel(
		int *dim, 
		int *ptr, 
		int *indices, 
		int *data,
		int *B,
		int *C)
{
	int tid = threadIdx.x;
	C[tid] = B[tid];
}

int main(int argc, char const *argv[])
{
	string file = "input1.txt";
	parse_input(file);
	init();

	kernel<<<1,dim>>>(d_dim, d_ptr, d_indices, d_data, d_B, d_C);

	anti_init();
	for(int i=0; i<C.size(); ++i) cout << C[i] << " "; cout << endl;
	return 0;
}